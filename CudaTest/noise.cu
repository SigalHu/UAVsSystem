#include "hip/hip_runtime.h"
﻿
#include "dev_noise.cuh"

#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>

bool cudaNoiseGene(float *noise_I, float *noise_Q, size_t length, float mean, float stddev){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_rand = NULL;
	
	try{
		hiprandGenerator_t gen;
		hiprandStatus_t cuRandStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		// hu 分配空间
		cudaStatus = hipMalloc((void **)&dev_rand, 2 * length*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandGenerateNormal(gen, dev_rand, 2 * length, mean, stddev);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_rand, length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)(dev_rand + length), length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch(bool &msg){
		isSucceed = msg;
	}

	if (dev_rand)
		hipFree(dev_rand);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}

bool cudaNoiseGeneWithSoS(float *noise_I, float *noise_Q,float fs, float time_spend, 
	float power_avg, unsigned int path_num, float fd_max, float delta_omega){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_omega_n_I = NULL, 
		*dev_omega_n_Q = NULL, 
		*dev_phi_n = NULL,
		*dev_cos_value = NULL,
		*dev_sin_value = NULL;

	try{
		hiprandStatus_t cuRandStatus;
		hiprandGenerator_t gen;
		size_t blockNum, threadNum;
		dim3 blockNum2D, threadNum2D;
		size_t col_num = fs*time_spend;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = path_num <= THREADS_EACH_BLOCK ? path_num : THREADS_EACH_BLOCK;
		blockNum = path_num % threadNum ?
			path_num / threadNum + 1 :
			path_num / threadNum;

		cudaStatus = hipMalloc((void **)&dev_omega_n_I, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMalloc((void **)&dev_omega_n_Q, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		noiseOmegaCulc<<<blockNum, threadNum>>>(dev_omega_n_I, dev_omega_n_Q, path_num,
			2 * M_PI*fd_max, (2 * M_PI - 2 * M_PI / (path_num + 1)) / (path_num - 1), delta_omega);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMalloc((void **)&dev_phi_n, 2*path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandGenerateUniform(gen, dev_phi_n, 2 * path_num);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		threadNum2D.x = BLOCK_DIM_X_32;
		threadNum2D.y = THREADS_EACH_BLOCK / BLOCK_DIM_X_32;
		blockNum2D.x = col_num % threadNum2D.x ? 
			col_num / threadNum2D.x + 1 : 
			col_num / threadNum2D.x;
		if (blockNum2D.x > GRID_DIM_LIMIT){
			unsigned int gridNum = blockNum2D.x % GRID_DIM_LIMIT ?
				blockNum2D.x / GRID_DIM_LIMIT + 1 :
				blockNum2D.x / GRID_DIM_LIMIT;
			blockNum2D.x = blockNum2D.x % gridNum ?
				blockNum2D.x / gridNum + 1 :
				blockNum2D.x / gridNum;
		}
		blockNum2D.y = path_num % threadNum2D.y ?
			path_num / threadNum2D.y + 1 :
			path_num / threadNum2D.y;

		size_t pitch;
		cudaStatus = hipMallocPitch((void **)&dev_cos_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMallocPitch((void **)&dev_sin_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		noiseSoSCulc<<<blockNum2D, threadNum2D>>>(dev_cos_value, dev_sin_value, pitch / sizeof(float), 
			col_num, path_num, 1 / fs, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = col_num <= THREADS_EACH_BLOCK ? col_num : THREADS_EACH_BLOCK;
		blockNum = col_num % threadNum ?
			col_num / threadNum + 1 :
			col_num / threadNum;
		if (blockNum > GRID_DIM_LIMIT){
			unsigned int gridNum = blockNum % GRID_DIM_LIMIT ?
				blockNum / GRID_DIM_LIMIT + 1 :
				blockNum / GRID_DIM_LIMIT;
			blockNum = blockNum % gridNum ?
				blockNum / gridNum + 1 :
				blockNum / gridNum;
		}
		noiseSoSSum<<<blockNum, threadNum >>>(dev_cos_value, dev_sin_value, pitch / sizeof(float),
			col_num, blockNum2D.y, sqrtf(power_avg / path_num));
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_cos_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)dev_sin_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_omega_n_I)
		hipFree(dev_omega_n_I);
	if (dev_omega_n_Q)
		hipFree(dev_omega_n_Q);
	if (dev_phi_n)
		hipFree(dev_phi_n);
	if (dev_cos_value)
		hipFree(dev_cos_value);
	if (dev_sin_value)
		hipFree(dev_sin_value);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
	    return false;
	}
	return isSucceed;
}

bool cudaNoiseGeneWithSoSBaseCol(float *noise_I, float *noise_Q, float fs, float time_spend,
	float power_avg, unsigned int path_num, float fd_max, float delta_omega){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_omega_n_I = NULL,
		*dev_omega_n_Q = NULL,
		*dev_phi_n = NULL,
		*dev_cos_value = NULL,
		*dev_sin_value = NULL;

	try{
		hiprandStatus_t cuRandStatus;
		hiprandGenerator_t gen;
		size_t blockNum, threadNum;
		dim3 blockNum2D, threadNum2D;
		size_t time_num = fs*time_spend;
		unsigned int time_num_eachrow, time_num_eachblock;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = path_num <= THREADS_EACH_BLOCK ? path_num : THREADS_EACH_BLOCK;
		blockNum = path_num % threadNum ?
			path_num / threadNum + 1 :
			path_num / threadNum;

		cudaStatus = hipMalloc((void **)&dev_omega_n_I, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMalloc((void **)&dev_omega_n_Q, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		noiseOmegaCulc<<<blockNum, threadNum>>>(dev_omega_n_I, dev_omega_n_Q, path_num,
			2 * M_PI*fd_max, (2 * M_PI - 2 * M_PI / (path_num + 1)) / (path_num - 1), delta_omega);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMalloc((void **)&dev_phi_n, 2 * path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandGenerateUniform(gen, dev_phi_n, 2 * path_num);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		if (path_num < BLOCK_DIM_X_32){
			time_num_eachrow = BLOCK_DIM_X_32 / path_num;
			threadNum2D.x = BLOCK_DIM_X_32;
		}
		else{
			time_num_eachrow = 1;
			if (path_num < BLOCK_DIM_X_64){
				threadNum2D.x = BLOCK_DIM_X_32;
			}
			else if (path_num < BLOCK_DIM_X_128){
				threadNum2D.x = BLOCK_DIM_X_64;
			}
			else if (path_num < BLOCK_DIM_X_256){
				threadNum2D.x = BLOCK_DIM_X_128;
			}
			else{
				threadNum2D.x = BLOCK_DIM_X_256;
			}
		}
		threadNum2D.y = THREADS_EACH_BLOCK / threadNum2D.x;
		time_num_eachblock = time_num_eachrow*threadNum2D.y;
		blockNum2D.x = 1;
		blockNum2D.y = time_num%time_num_eachblock ? 
			time_num / time_num_eachblock + 1 : 
			time_num / time_num_eachblock;
		if (blockNum2D.y > GRID_DIM_LIMIT){
			unsigned int gridNum = blockNum2D.y % GRID_DIM_LIMIT ?
				blockNum2D.y / GRID_DIM_LIMIT + 1 :
				blockNum2D.y / GRID_DIM_LIMIT;
			blockNum2D.y = blockNum2D.y % gridNum ?
				blockNum2D.y / gridNum + 1 :
				blockNum2D.y / gridNum;
		}

		cudaStatus = hipMalloc((void **)&dev_cos_value, time_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMalloc((void **)&dev_sin_value, time_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		//noiseSoSCulc<<<blockNum2D, threadNum2D>>>(dev_cos_value, dev_sin_value, pitch / sizeof(float),
		//	col_num, path_num, 1 / fs, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) {
		//	throw false;
		//}

		//threadNum = col_num <= threadNumLimit ? col_num : threadNumLimit;
		//blockNum = col_num % threadNum ?
		//	col_num / threadNum + 1 :
		//	col_num / threadNum;
		//if (blockNum > GRID_DIM_LIMIT){
		//	unsigned int gridNum = blockNum % GRID_DIM_LIMIT ?
		//		blockNum / GRID_DIM_LIMIT + 1 :
		//		blockNum / GRID_DIM_LIMIT;
		//	blockNum = blockNum % gridNum ?
		//		blockNum / gridNum + 1 :
		//		blockNum / gridNum;
		//}
		//noiseSoSSum << <blockNum, threadNum >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
		//	col_num, blockNum2D.y, sqrtf(power_avg / path_num));
		//cudaStatus = hipGetLastError();
		//if (cudaStatus != hipSuccess) {
		//	throw false;
		//}

		//cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_cos_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		//if (cudaStatus != hipSuccess) {
		//	throw false;
		//}
		//cudaStatus = hipMemcpy((void *)noise_Q, (void *)dev_sin_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		//if (cudaStatus != hipSuccess) {
		//	throw false;
		//}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_omega_n_I)
		hipFree(dev_omega_n_I);
	if (dev_omega_n_Q)
		hipFree(dev_omega_n_Q);
	if (dev_phi_n)
		hipFree(dev_phi_n);
	if (dev_cos_value)
		hipFree(dev_cos_value);
	if (dev_sin_value)
		hipFree(dev_sin_value);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}