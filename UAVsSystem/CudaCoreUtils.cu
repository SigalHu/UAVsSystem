#include "common.h"
#include "hip/hip_runtime.h"
#include "CudaCoreUtils.h"

const dim3 CudaCoreUtils::DEFAULT_GRID_DIM(65535, 65535, 65535);
std::map<unsigned int, const dim3> CudaCoreUtils::gridDimMap = CudaCoreUtils::getGridDimList();

std::string CudaCoreUtils::getClassName(){
	return MacroUtils_ClassName(CudaCoreUtils);
}

std::map<unsigned int, const dim3> CudaCoreUtils::getGridDimList(){
	int count = 0;
	hipDeviceProp_t prop;
	std::map<unsigned int, const dim3> map;

	hipError_t error = hipGetDeviceCount(&count);
	if (error != hipSuccess)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hipGetDeviceCount), hipGetErrorString(error));

	for (unsigned int ii = 0; ii < count; ++ii){
		if (hipGetDeviceProperties(&prop, ii) != hipSuccess)
			throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
			getClassName(), MacroUtils_CurFunctionName(),
			MacroUtils_FunctionName(hipGetDeviceProperties), hipGetErrorString(error));
		else
			map.emplace(ii, dim3(prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]));
	}
	return map;
}

const dim3 CudaCoreUtils::getGridDim(const unsigned int &deviceId){
	std::map<unsigned int, const dim3>::iterator iter = gridDimMap.find(deviceId);
	if (iter == gridDimMap.end()){
		return DEFAULT_GRID_DIM;
	}
	return iter->second;
}

void CudaCoreUtils::setDevice(int device){
	hipError_t error = hipSetDevice(device);
	if (hipSuccess != error)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hipSetDevice), hipGetErrorString(error));
}

void CudaCoreUtils::resetDevice(){
	hipError_t error = hipDeviceReset();
	if (hipSuccess != error)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hipDeviceReset), hipGetErrorString(error));
}

int CudaCoreUtils::getDeviceCount(){
	int count = 0;
	hipError_t error = hipGetDeviceCount(&count);
	if (hipSuccess != error)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hipGetDeviceCount), hipGetErrorString(error));
	return count;
}

void CudaCoreUtils::getDeviceProperties(int device, hipDeviceProp_t &prop){
	hipError_t error = hipGetDeviceProperties(&prop, device);
	if (hipSuccess != error)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hipGetDeviceProperties), hipGetErrorString(error));
}

void CudaCoreUtils::printDeviceProperties(){
	int count = getDeviceCount();
	hipDeviceProp_t prop;
	for (int ii = 0; ii < count; ii++){
		getDeviceProperties(ii, prop);
		printf("###############################################\n");
		printf("Device Name : %s.\n", prop.name);
		printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
		printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
		printf("regsPerBlock : %d.\n", prop.regsPerBlock);
		printf("warpSize : %d.\n", prop.warpSize);
		printf("memPitch : %d.\n", prop.memPitch);
		printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
		printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("totalConstMem : %d.\n", prop.totalConstMem);
		printf("major.minor : %d.%d.\n", prop.major, prop.minor);
		printf("clockRate : %d.\n", prop.clockRate);
		printf("textureAlignment : %d.\n", prop.textureAlignment);
		printf("deviceOverlap : %d.\n", prop.deviceOverlap);
		printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
		printf("###############################################\n");
	}
}