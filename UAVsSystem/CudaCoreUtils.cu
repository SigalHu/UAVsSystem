#include "CudaCoreUtils.h"

bool CudaCoreUtils::malloc(void **devPtr, size_t size){
	return hipSuccess == hipMalloc(devPtr, size);
}

bool CudaCoreUtils::mallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height){
	return hipSuccess == hipMallocPitch(devPtr, pitch, width, height);
}

void CudaCoreUtils::free(void *devPtr){
	if (devPtr)
		hipFree(devPtr);
}

bool CudaCoreUtils::memcpyDeviceToHost(void *dst, const void *src, size_t count){
	return hipSuccess == hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}