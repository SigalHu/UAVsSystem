#include "CudaCommonUtils.h"

bool CudaCommonUtils::randCreateGenerator(hiprandGenerator_t *generator, hiprandRngType_t rng_type){
	if (HIPRAND_STATUS_SUCCESS != hiprandCreateGenerator(generator, rng_type))
		return false;
	return HIPRAND_STATUS_SUCCESS == hiprandSetPseudoRandomGeneratorSeed(*generator, time(NULL));
}

bool CudaCommonUtils::randGenerateNormal(float *outputPtr, size_t n, float mean, float stddev){
	hiprandGenerator_t generator;
	if (!randCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT))
		return false;
	return HIPRAND_STATUS_SUCCESS == hiprandGenerateNormal(generator, outputPtr, n, mean, stddev);
}

bool CudaCommonUtils::randGenerateUniform(float *outputPtr, size_t num){
	hiprandGenerator_t generator;
	if (!randCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT))
		return false;
	return HIPRAND_STATUS_SUCCESS == hiprandGenerateUniform(generator, outputPtr, num);
}

bool CudaCommonUtils::cudaNoiseGene(float *noise_I, float *noise_Q, size_t length, float mean, float stddev){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_rand = NULL;

	try{
		hiprandGenerator_t gen;
		hiprandStatus_t cuRandStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		// hu ����ռ�
		cudaStatus = hipMalloc((void **)&dev_rand, 2 * length*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandGenerateNormal(gen, dev_rand, 2 * length, mean, stddev);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_rand, length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)(dev_rand + length), length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_rand)
		hipFree(dev_rand);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}