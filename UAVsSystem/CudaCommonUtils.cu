#include "CudaCommonUtils.h"

bool CudaCommonUtils::cudaNoiseGene(float *noise_I, float *noise_Q, size_t length, float mean, float stddev){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_rand = NULL;

	try{
		hiprandGenerator_t gen;
		hiprandStatus_t cuRandStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		// hu ����ռ�
		cudaStatus = hipMalloc((void **)&dev_rand, 2 * length*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandGenerateNormal(gen, dev_rand, 2 * length, mean, stddev);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_rand, length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)(dev_rand + length), length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_rand)
		hipFree(dev_rand);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}