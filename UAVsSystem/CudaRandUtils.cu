#include <time.h>
#include <string>
#include "common.h"
#include "DeviceVector.hpp"
#include "CudaRandUtils.h"

std::string CudaRandUtils::getClassName(){
	return MacroUtils_ClassName(CudaRandUtils);
}

std::string CudaRandUtils::getStatusStr(hiprandStatus_t status){
	switch (status){
	case HIPRAND_STATUS_SUCCESS:
		return "No errors.";
	case HIPRAND_STATUS_VERSION_MISMATCH:
		return "Header file and linked library version do not match.";
	case HIPRAND_STATUS_NOT_INITIALIZED:
		return "Generator not initialized.";
	case HIPRAND_STATUS_ALLOCATION_FAILED:
		return "Memory allocation failed.";
	case HIPRAND_STATUS_TYPE_ERROR:
		return "Generator is wrong type.";
	case HIPRAND_STATUS_OUT_OF_RANGE:
		return "Argument out of range.";
	case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
		return "Length requested is not a multple of dimension.";
	case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
		return "GPU does not have double precision required by MRG32k3a.";
	case HIPRAND_STATUS_LAUNCH_FAILURE:
		return "Kernel launch failure.";
	case HIPRAND_STATUS_PREEXISTING_FAILURE:
		return "Preexisting failure on library entry.";
	case HIPRAND_STATUS_INITIALIZATION_FAILED:
		return "Initialization of CUDA failed.";
	case HIPRAND_STATUS_ARCH_MISMATCH:
		return "Architecture mismatch, GPU does not support requested feature.";
	case HIPRAND_STATUS_INTERNAL_ERROR:
		return "Internal library error.";
	default:
		return "unrecognized error code.";
	}
}

void CudaRandUtils::createGenerator(hiprandGenerator_t &generator, hiprandRngType_t rng_type){
	hiprandStatus_t status = hiprandCreateGenerator(&generator, rng_type);
	if (HIPRAND_STATUS_SUCCESS != status)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hiprandCreateGenerator), getStatusStr(status));

	status = hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));
	if (HIPRAND_STATUS_SUCCESS != status)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hiprandSetPseudoRandomGeneratorSeed), getStatusStr(status));
}

void CudaRandUtils::generateNormal(DeviceVector<float> &vector, float mean, float stddev){
	hiprandGenerator_t generator;
	createGenerator(generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandStatus_t status = hiprandGenerateNormal(generator, raw_pointer_cast(vector.data()), vector.size(), mean, stddev);
	if (HIPRAND_STATUS_SUCCESS != status)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hiprandGenerateNormal), getStatusStr(status));
}

void CudaRandUtils::generateUniform(DeviceVector<float> &vector){
	hiprandGenerator_t generator;
	createGenerator(generator, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandStatus_t status = hiprandGenerateUniform(generator, raw_pointer_cast(vector.data()), vector.size());
	if (HIPRAND_STATUS_SUCCESS != status)
		throw SystemException(SystemCodeEnum::CUDA_RUNTIME_ERROR,
		getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(hiprandGenerateUniform), getStatusStr(status));
}

bool CudaRandUtils::cudaNoiseGene(float *noise_I, float *noise_Q, size_t length, float mean, float stddev){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_rand = NULL;

	try{
		hiprandGenerator_t gen;
		hiprandStatus_t cuRandStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		// hu ����ռ�
		cudaStatus = hipMalloc((void **)&dev_rand, 2 * length*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cuRandStatus = hiprandGenerateNormal(gen, dev_rand, 2 * length, mean, stddev);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_rand, length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)(dev_rand + length), length*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_rand)
		hipFree(dev_rand);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}