#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include ""
#include "hiprand.h"
#include <math.h>
#include <time.h>
#include "common.h"
#include "cuda_definition.h"
#include "cuda_api.h"
#include "CudaSoSUtils.h"

std::string CudaSoSUtils::getClassName(){
	return MacroUtils_ClassName(CudaSoSUtils);
}

template<class _Alloc>
void CudaSoSUtils::noiseGene(DeviceVector<float, _Alloc> &noiseI, DeviceVector<float, _Alloc> &noiseQ, const float &fs,
	const float &avgPower, const unsigned int &pathNum, const float &maxFd, const float &deltaOmega){
	if (noiseI.empty() || noiseQ.empty())
		throw SystemException(SystemCodeEnum::OUT_OF_RANGE, getClassName(), MacroUtils_CurFunctionName(), MacroUtils_VariableName(noiseI),
		MacroUtils_VariableName(noiseI).append(" and ").append(MacroUtils_VariableName(noiseQ)).append(" can not be empty."));
	if (noiseI.size() != noiseQ.size())
		throw SystemException(SystemCodeEnum::NOT_EQUAL, getClassName(), MacroUtils_CurFunctionName(), MacroUtils_VariableName(noiseI),
		StringUtils::format(MacroUtils_VariableName(noiseI).append("(size=%d) is not equal to ").append(MacroUtils_VariableName(noiseQ)).append("(size=%d)."),
		noiseI.size(), noiseQ.size()));
	unsigned int tmp = 1;
	while (tmp < pathNum)
		tmp <<= 1;
	if (pathNum != tmp)
		throw SystemException(SystemCodeEnum::NOT_EQUAL, getClassName(), MacroUtils_CurFunctionName(), MacroUtils_VariableName(pathNum),
		StringUtils::format(MacroUtils_VariableName(pathNum).append("(=%d) is not equal to pow(2,n)."), pathNum));

	float omegaAmp = 2 * M_PI*maxFd;
	float deltaAlpha = (2 * M_PI - 2 * M_PI / (pathNum + 1)) / (pathNum - 1);
	float deltaT = 1 / fs;
	float sumAmp = sqrtf(avgPower / pathNum);

	size_t len = noiseI.size();
	dim3 blockNum, threadNum(THREAD_NUM_PER_BLOCK);
	if (pathNum <= THREAD_NUM_PER_BLOCK){
		blockNum.y = len;
	}
	else{
		blockNum.x = pathNum / THREAD_NUM_PER_BLOCK;
		blockNum.y = len / blockNum.x + (len % blockNum.x) != 0;
	}
	if (blockNum.y > gridDim.y){
		blockNum.y = gridDim.y;
	}

	cudaNoiseGeneWithSoS << <blockNum, threadNum, 2 * pathNum*sizeof(float) >> >(
		raw_pointer_cast(noiseI.data()), raw_pointer_cast(noiseQ.data()),
		len, pathNum, time(NULL), omegaAmp, deltaAlpha, deltaOmega, deltaT, sumAmp);

	hipError_t error = hipGetLastError();
	if (hipSuccess != error)
		throw SystemException(SystemCodeEnum::CUDA_CALL_ERROR, getClassName(), MacroUtils_CurFunctionName(),
		MacroUtils_FunctionName(cudaNoiseGeneWithSoS), hipGetErrorString(error));
}
//
//template<class _Alloc>
//void CudaSoSUtils::noiseOmegaCulc(const dim3 &blockNum, const dim3 &threadNum,
//	DeviceVector<float, _Alloc> &devOmegaI, DeviceVector<float, _Alloc> &devOmegaQ,
//	const float &maxFd, const float &deltaOmega){
//	if (devOmegaI.empty() || devOmegaQ.empty())
//		throw SystemException(SystemCodeEnum::OUT_OF_RANGE, MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaI),
//		MacroUtils_VariableName(devOmegaI).append(" and ").append(MacroUtils_VariableName(devOmegaQ)).append(" can not be empty."));
//	if (devOmegaI.size() != devOmegaQ.size())
//		throw SystemException(SystemCodeEnum::NOT_EQUAL, MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaI),
//		StringUtils::format(MacroUtils_VariableName(devOmegaI).append("(size=%d) is not equal to ").append(MacroUtils_VariableName(devOmegaQ)).append("(size=%d)."),
//		devOmegaI.size(), devOmegaQ.size()));
//	
//	unsigned int pathNum = devOmegaI.size();
//	cudaNoiseOmegaCulc << <blockNum, threadNum >> >(raw_pointer_cast(devOmegaI.data()), raw_pointer_cast(devOmegaQ.data()),
//		pathNum, 2 * M_PI*maxFd, (2 * M_PI - 2 * M_PI / (pathNum + 1)) / (pathNum - 1), deltaOmega);
//
//	hipError_t error = hipGetLastError();
//	if (hipSuccess != error)
//		throw SystemException(SystemCodeEnum::CUDA_CALL_ERROR, MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(),
//		MacroUtils_FunctionName(cudaNoiseOmegaCulc), hipGetErrorString(error));
//}
//
//bool CudaSoSUtils::noiseSoSCulc(float* const &devCosValue, float* const &devSinValue, const dim3 &blockNum, const dim3 &threadNum,
//	const unsigned int &pitchWidth, const unsigned int &width, const unsigned int &heigth, const float &deltaT,
//	float* const &devOmegaI, float* const &devOmegaQ, float* const &devPhi) {
//	//if (devCosValue == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devCosValue), "null."));
//	//if (devSinValue == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devSinValue), "null."));
//	//if (devOmegaI == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaI), "null."));
//	//if (devOmegaQ == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaQ), "null."));
//	//if (devPhi == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devPhi), "null."));
//
//	cudaNoiseSoSCulc << <blockNum, threadNum >> >(devCosValue, devSinValue, 
//		pitchWidth, width, heigth, deltaT, devOmegaI, devOmegaQ, devPhi);
//	return hipSuccess == hipGetLastError();
//}
//
//bool CudaSoSUtils::noiseSoSSum(float* const &devCosValue, float* const &devSinValue, const dim3 &blockNum, const dim3 &threadNum,
//	const unsigned int &pitchWidth, const unsigned int &width, const unsigned int &heigth, const float &sumAmp){
//	//if (devCosValue == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devCosValue), "null."));
//	//if (devSinValue == nullptr)
//	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
//	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
//	//		MacroUtils_ClassName(CudaSoSUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devSinValue), "null."));
//
//	cudaNoiseSoSSum << <blockNum, threadNum >> >(devCosValue, devSinValue, pitchWidth, width, heigth, sumAmp);
//	return hipSuccess == hipGetLastError();
//}
//
//bool CudaSoSUtils::cudaNoiseGeneWithSoS(float *noise_I, float *noise_Q, float fs, float time_spend,
//	float power_avg, unsigned int path_num, float fd_max, float delta_omega){
//	bool isSucceed = true;
//	hipError_t cudaStatus;
//	float *dev_omega_n_I = NULL,
//		*dev_omega_n_Q = NULL,
//		*dev_phi_n = NULL,
//		*dev_cos_value = NULL,
//		*dev_sin_value = NULL;
//
//	try{
//		hiprandStatus_t cuRandStatus;
//		hiprandGenerator_t gen;
//		const unsigned int threadNumLimit = 256;
//		size_t blockNum, threadNum;
//		dim3 blockNum2D, threadNum2D;
//		size_t col_num = fs*time_spend;
//
//		cudaStatus = hipSetDevice(0);
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		threadNum = path_num <= threadNumLimit ? path_num : threadNumLimit;
//		blockNum = path_num % threadNum ?
//			path_num / threadNum + 1 :
//			path_num / threadNum;
//
//		cudaStatus = hipMalloc((void **)&dev_omega_n_I, path_num*sizeof(float));
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//		cudaStatus = hipMalloc((void **)&dev_omega_n_Q, path_num*sizeof(float));
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		cudaNoiseOmegaCulc << <blockNum, threadNum >> >(dev_omega_n_I, dev_omega_n_Q, path_num,
//			2 * M_PI*fd_max, (2 * M_PI - 2 * M_PI / (path_num + 1)) / (path_num - 1), delta_omega);
//		cudaStatus = hipGetLastError();
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		cudaStatus = hipMalloc((void **)&dev_phi_n, 2 * path_num*sizeof(float));
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
//		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
//			throw false;
//		}
//		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
//		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
//			throw false;
//		}
//		cuRandStatus = hiprandGenerateUniform(gen, dev_phi_n, 2 * path_num);
//		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
//			throw false;
//		}
//
//		threadNum2D.x = BLOCK_DIM_X;
//		threadNum2D.y = BLOCK_DIM_Y;
//		blockNum2D.x = col_num % threadNum2D.x ?
//			col_num / threadNum2D.x + 1 :
//			col_num / threadNum2D.x;
//		if (blockNum2D.x > GRID_DIM_X){
//			unsigned int gridNum = blockNum2D.x % GRID_DIM_X ?
//				blockNum2D.x / GRID_DIM_X + 1 :
//				blockNum2D.x / GRID_DIM_X;
//			blockNum2D.x = blockNum2D.x % gridNum ?
//				blockNum2D.x / gridNum + 1 :
//				blockNum2D.x / gridNum;
//		}
//		blockNum2D.y = path_num % threadNum2D.y ?
//			path_num / threadNum2D.y + 1 :
//			path_num / threadNum2D.y;
//
//
//		size_t pitch;
//		cudaStatus = hipMallocPitch((void **)&dev_cos_value, &pitch, col_num*sizeof(float), blockNum2D.y);
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//		cudaStatus = hipMallocPitch((void **)&dev_sin_value, &pitch, col_num*sizeof(float), blockNum2D.y);
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//		cudaNoiseSoSCulc << <blockNum2D, threadNum2D >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
//			col_num, path_num, 1 / fs, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
//		cudaStatus = hipGetLastError();
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		threadNum = col_num <= threadNumLimit ? col_num : threadNumLimit;
//		blockNum = col_num % threadNum ?
//			col_num / threadNum + 1 :
//			col_num / threadNum;
//		if (blockNum > GRID_DIM_X){
//			unsigned int gridNum = blockNum % GRID_DIM_X ?
//				blockNum / GRID_DIM_X + 1 :
//				blockNum / GRID_DIM_X;
//			blockNum = blockNum % gridNum ?
//				blockNum / gridNum + 1 :
//				blockNum / gridNum;
//		}
//		cudaNoiseSoSSum << <blockNum, threadNum >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
//			col_num, blockNum2D.y, sqrtf(power_avg / path_num));
//		cudaStatus = hipGetLastError();
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//
//		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_cos_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//		cudaStatus = hipMemcpy((void *)noise_Q, (void *)dev_sin_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
//		if (cudaStatus != hipSuccess) {
//			throw false;
//		}
//	}
//	catch (bool &msg){
//		isSucceed = msg;
//	}
//
//	if (dev_omega_n_I)
//		hipFree(dev_omega_n_I);
//	if (dev_omega_n_Q)
//		hipFree(dev_omega_n_Q);
//	if (dev_phi_n)
//		hipFree(dev_phi_n);
//	if (dev_cos_value)
//		hipFree(dev_cos_value);
//	if (dev_sin_value)
//		hipFree(dev_sin_value);
//
//	cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		return false;
//	}
//	return isSucceed;
//}
