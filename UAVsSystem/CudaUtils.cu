#include "CudaUtils.h"

string CudaUtils::lastError = "";

const string& CudaUtils::getLastError() const{
	return CudaUtils::lastError;
}

bool CudaUtils::setDevice(int device){
	hipError_t error = hipSetDevice(device);
	CudaUtils::lastError = hipGetErrorString(error);
	return hipSuccess == error;
}

bool CudaUtils::resetDevice(){
	hipError_t error = hipDeviceReset();
	CudaUtils::lastError = hipGetErrorString(error);
	return hipSuccess == error;
}

int CudaUtils::getDeviceCount(){
	int count = 0;
	hipError_t error = hipGetDeviceCount(&count);
	CudaUtils::lastError = hipGetErrorString(error);
	return count;
}

bool CudaUtils::getDeviceProperties(int device, hipDeviceProp_t &prop){
	hipError_t error = hipGetDeviceProperties(&prop, device);
	CudaUtils::lastError = hipGetErrorString(error);
	return hipSuccess == error;
}

void CudaUtils::printDeviceProperties(){
	int count = getDeviceCount();
	hipDeviceProp_t prop;
	for (int ii = 0; ii < count; ii++){
		if (getDeviceProperties(ii, prop)){
			printf("###############################################\n");
			printf("Device Name : %s.\n", prop.name);
			printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
			printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
			printf("regsPerBlock : %d.\n", prop.regsPerBlock);
			printf("warpSize : %d.\n", prop.warpSize);
			printf("memPitch : %d.\n", prop.memPitch);
			printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
			printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
			printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("totalConstMem : %d.\n", prop.totalConstMem);
			printf("major.minor : %d.%d.\n", prop.major, prop.minor);
			printf("clockRate : %d.\n", prop.clockRate);
			printf("textureAlignment : %d.\n", prop.textureAlignment);
			printf("deviceOverlap : %d.\n", prop.deviceOverlap);
			printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
			printf("###############################################\n");
		}
	}
}