#include "CudaUtils.h"

bool CudaUtils::setDevice(int device){
	return hipSuccess == hipSetDevice(device);
}

bool CudaUtils::resetDevice(){
	return hipSuccess == hipDeviceReset();
}

int CudaUtils::getDeviceCount(){
	int count = 0;
	hipGetDeviceCount(&count);
	return count;
}

bool CudaUtils::getDeviceProperties(int device, hipDeviceProp_t *prop){
	return hipSuccess == hipGetDeviceProperties(prop, device);
}

void CudaUtils::printDeviceProperties(){
	int count = getDeviceCount();
	hipDeviceProp_t prop;
	for (int ii = 0; ii < count; ii++){
		if (getDeviceProperties(ii, &prop)){
			printf("###############################################\n");
			printf("Device Name : %s.\n", prop.name);
			printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
			printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
			printf("regsPerBlock : %d.\n", prop.regsPerBlock);
			printf("warpSize : %d.\n", prop.warpSize);
			printf("memPitch : %d.\n", prop.memPitch);
			printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
			printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
			printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
			printf("totalConstMem : %d.\n", prop.totalConstMem);
			printf("major.minor : %d.%d.\n", prop.major, prop.minor);
			printf("clockRate : %d.\n", prop.clockRate);
			printf("textureAlignment : %d.\n", prop.textureAlignment);
			printf("deviceOverlap : %d.\n", prop.deviceOverlap);
			printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
			printf("###############################################\n");
		}
	}
}