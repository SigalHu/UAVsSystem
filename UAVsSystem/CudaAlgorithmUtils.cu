#include "hip/hip_runtime.h"
#include "CudaAlgorithmUtils.h"

template<size_t _Size>
void CudaAlgorithmUtils::noiseOmegaCulc(ComplexArray<float, _Size> &devOmega, const unsigned int &pathNum, const float &maxFd, const float &deltaOmega){
	size_t blockNum, threadNum;

	//cudaNoiseOmegaCulc(dev_omega_n_I, dev_omega_n_Q, tid_max, omega_amp, delta_alpha, delta_omega);
}

void CudaAlgorithmUtils::noiseSoSCulc(float *dev_cos_value, float *dev_sin_value,
	unsigned int pitch_width, unsigned int width, unsigned int heigth, float delta_t,
	float *dev_omega_n_I, float *dev_omega_n_Q, float *dev_phi_n){
	//cudaNoiseSoSCulc(dev_cos_value, dev_sin_value, pitch_width, width, heigth, delta_t, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
}

void CudaAlgorithmUtils::noiseSoSSum(float *dev_cos_value, float *dev_sin_value,
	unsigned int pitch_width, unsigned int width, unsigned int heigth, float sum_amp){
	//cudaNoiseSoSSum(dev_cos_value, dev_sin_value, pitch_width, width, heigth, sum_amp);
}

bool CudaAlgorithmUtils::cudaNoiseGeneWithSoS(float *noise_I, float *noise_Q, float fs, float time_spend,
	float power_avg, unsigned int path_num, float fd_max, float delta_omega){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_omega_n_I = NULL,
		*dev_omega_n_Q = NULL,
		*dev_phi_n = NULL,
		*dev_cos_value = NULL,
		*dev_sin_value = NULL;

	try{
		hiprandStatus_t cuRandStatus;
		hiprandGenerator_t gen;
		const unsigned int threadNumLimit = 256;
		size_t blockNum, threadNum;
		dim3 blockNum2D, threadNum2D;
		size_t col_num = fs*time_spend;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = path_num <= threadNumLimit ? path_num : threadNumLimit;
		blockNum = path_num % threadNum ?
			path_num / threadNum + 1 :
			path_num / threadNum;

		cudaStatus = hipMalloc((void **)&dev_omega_n_I, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMalloc((void **)&dev_omega_n_Q, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaNoiseOmegaCulc << <blockNum, threadNum >> >(dev_omega_n_I, dev_omega_n_Q, path_num,
			2 * M_PI*fd_max, (2 * M_PI - 2 * M_PI / (path_num + 1)) / (path_num - 1), delta_omega);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMalloc((void **)&dev_phi_n, 2 * path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandGenerateUniform(gen, dev_phi_n, 2 * path_num);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		threadNum2D.x = BLOCK_DIM_X;
		threadNum2D.y = BLOCK_DIM_Y;
		blockNum2D.x = col_num % threadNum2D.x ?
			col_num / threadNum2D.x + 1 :
			col_num / threadNum2D.x;
		if (blockNum2D.x > GRID_DIM_X){
			unsigned int gridNum = blockNum2D.x % GRID_DIM_X ?
				blockNum2D.x / GRID_DIM_X + 1 :
				blockNum2D.x / GRID_DIM_X;
			blockNum2D.x = blockNum2D.x % gridNum ?
				blockNum2D.x / gridNum + 1 :
				blockNum2D.x / gridNum;
		}
		blockNum2D.y = path_num % threadNum2D.y ?
			path_num / threadNum2D.y + 1 :
			path_num / threadNum2D.y;


		size_t pitch;
		cudaStatus = hipMallocPitch((void **)&dev_cos_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMallocPitch((void **)&dev_sin_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaNoiseSoSCulc << <blockNum2D, threadNum2D >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
			col_num, path_num, 1 / fs, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = col_num <= threadNumLimit ? col_num : threadNumLimit;
		blockNum = col_num % threadNum ?
			col_num / threadNum + 1 :
			col_num / threadNum;
		if (blockNum > GRID_DIM_X){
			unsigned int gridNum = blockNum % GRID_DIM_X ?
				blockNum / GRID_DIM_X + 1 :
				blockNum / GRID_DIM_X;
			blockNum = blockNum % gridNum ?
				blockNum / gridNum + 1 :
				blockNum / gridNum;
		}
		cudaNoiseSoSSum << <blockNum, threadNum >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
			col_num, blockNum2D.y, sqrtf(power_avg / path_num));
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_cos_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)dev_sin_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_omega_n_I)
		hipFree(dev_omega_n_I);
	if (dev_omega_n_Q)
		hipFree(dev_omega_n_Q);
	if (dev_phi_n)
		hipFree(dev_phi_n);
	if (dev_cos_value)
		hipFree(dev_cos_value);
	if (dev_sin_value)
		hipFree(dev_sin_value);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}
