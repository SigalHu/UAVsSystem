#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand/hiprand_kernel.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <time.h>
#include "cuda_api.h"
#include "common.h"
#include "CudaAlgorithmUtils.h"

bool CudaAlgorithmUtils::noiseOmegaCulc(float* const &devOmegaI, float* const &devOmegaQ, const dim3 &blockNum, const dim3 &threadNum,
	const unsigned int &pathNum, const float &maxFd, const float &deltaOmega){
	//if (pathNum == 0)
	//	throw SystemException(SystemCodeEnum::OUT_OF_RANGE,
	//		StringUtils::format(SystemCodeEnum::OUT_OF_RANGE.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(pathNum),
	//		MacroUtils_VariableName(pathNum) + " must be greater than 0."));
	//if (devOmegaI == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaI), "null."));
	//if (devOmegaQ == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaQ), "null."));

	cudaNoiseOmegaCulc << <blockNum, threadNum >> >(devOmegaI, devOmegaQ, pathNum,
		2 * M_PI*maxFd, (2 * M_PI - 2 * M_PI / (pathNum + 1)) / (pathNum - 1), deltaOmega);
	return hipSuccess == hipGetLastError();
}

bool CudaAlgorithmUtils::noiseSoSCulc(float* const &devCosValue, float* const &devSinValue, const dim3 &blockNum, const dim3 &threadNum,
	const unsigned int &pitchWidth, const unsigned int &width, const unsigned int &heigth, const float &deltaT,
	float* const &devOmegaI, float* const &devOmegaQ, float* const &devPhi) {
	//if (devCosValue == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devCosValue), "null."));
	//if (devSinValue == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devSinValue), "null."));
	//if (devOmegaI == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaI), "null."));
	//if (devOmegaQ == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devOmegaQ), "null."));
	//if (devPhi == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devPhi), "null."));

	cudaNoiseSoSCulc << <blockNum, threadNum >> >(devCosValue, devSinValue, 
		pitchWidth, width, heigth, deltaT, devOmegaI, devOmegaQ, devPhi);
	return hipSuccess == hipGetLastError();
}

bool CudaAlgorithmUtils::noiseSoSSum(float* const &devCosValue, float* const &devSinValue, const dim3 &blockNum, const dim3 &threadNum,
	const unsigned int &pitchWidth, const unsigned int &width, const unsigned int &heigth, const float &sumAmp){
	//if (devCosValue == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devCosValue), "null."));
	//if (devSinValue == nullptr)
	//	throw SystemException(SystemCodeEnum::NULL_POINTER,
	//		StringUtils::format(SystemCodeEnum::NULL_POINTER.getInfo(),
	//		MacroUtils_ClassName(CudaAlgorithmUtils), MacroUtils_CurFunctionName(), MacroUtils_VariableName(devSinValue), "null."));

	cudaNoiseSoSSum << <blockNum, threadNum >> >(devCosValue, devSinValue, pitchWidth, width, heigth, sumAmp);
	return hipSuccess == hipGetLastError();
}

bool CudaAlgorithmUtils::cudaNoiseGeneWithSoS(float *noise_I, float *noise_Q, float fs, float time_spend,
	float power_avg, unsigned int path_num, float fd_max, float delta_omega){
	bool isSucceed = true;
	hipError_t cudaStatus;
	float *dev_omega_n_I = NULL,
		*dev_omega_n_Q = NULL,
		*dev_phi_n = NULL,
		*dev_cos_value = NULL,
		*dev_sin_value = NULL;

	try{
		hiprandStatus_t cuRandStatus;
		hiprandGenerator_t gen;
		const unsigned int threadNumLimit = 256;
		size_t blockNum, threadNum;
		dim3 blockNum2D, threadNum2D;
		size_t col_num = fs*time_spend;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = path_num <= threadNumLimit ? path_num : threadNumLimit;
		blockNum = path_num % threadNum ?
			path_num / threadNum + 1 :
			path_num / threadNum;

		cudaStatus = hipMalloc((void **)&dev_omega_n_I, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMalloc((void **)&dev_omega_n_Q, path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaNoiseOmegaCulc << <blockNum, threadNum >> >(dev_omega_n_I, dev_omega_n_Q, path_num,
			2 * M_PI*fd_max, (2 * M_PI - 2 * M_PI / (path_num + 1)) / (path_num - 1), delta_omega);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMalloc((void **)&dev_phi_n, 2 * path_num*sizeof(float));
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cuRandStatus = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}
		cuRandStatus = hiprandGenerateUniform(gen, dev_phi_n, 2 * path_num);
		if (cuRandStatus != HIPRAND_STATUS_SUCCESS){
			throw false;
		}

		threadNum2D.x = BLOCK_DIM_X;
		threadNum2D.y = BLOCK_DIM_Y;
		blockNum2D.x = col_num % threadNum2D.x ?
			col_num / threadNum2D.x + 1 :
			col_num / threadNum2D.x;
		if (blockNum2D.x > GRID_DIM_X){
			unsigned int gridNum = blockNum2D.x % GRID_DIM_X ?
				blockNum2D.x / GRID_DIM_X + 1 :
				blockNum2D.x / GRID_DIM_X;
			blockNum2D.x = blockNum2D.x % gridNum ?
				blockNum2D.x / gridNum + 1 :
				blockNum2D.x / gridNum;
		}
		blockNum2D.y = path_num % threadNum2D.y ?
			path_num / threadNum2D.y + 1 :
			path_num / threadNum2D.y;


		size_t pitch;
		cudaStatus = hipMallocPitch((void **)&dev_cos_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMallocPitch((void **)&dev_sin_value, &pitch, col_num*sizeof(float), blockNum2D.y);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaNoiseSoSCulc << <blockNum2D, threadNum2D >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
			col_num, path_num, 1 / fs, dev_omega_n_I, dev_omega_n_Q, dev_phi_n);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		threadNum = col_num <= threadNumLimit ? col_num : threadNumLimit;
		blockNum = col_num % threadNum ?
			col_num / threadNum + 1 :
			col_num / threadNum;
		if (blockNum > GRID_DIM_X){
			unsigned int gridNum = blockNum % GRID_DIM_X ?
				blockNum / GRID_DIM_X + 1 :
				blockNum / GRID_DIM_X;
			blockNum = blockNum % gridNum ?
				blockNum / gridNum + 1 :
				blockNum / gridNum;
		}
		cudaNoiseSoSSum << <blockNum, threadNum >> >(dev_cos_value, dev_sin_value, pitch / sizeof(float),
			col_num, blockNum2D.y, sqrtf(power_avg / path_num));
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			throw false;
		}

		cudaStatus = hipMemcpy((void *)noise_I, (void *)dev_cos_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
		cudaStatus = hipMemcpy((void *)noise_Q, (void *)dev_sin_value, col_num*sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			throw false;
		}
	}
	catch (bool &msg){
		isSucceed = msg;
	}

	if (dev_omega_n_I)
		hipFree(dev_omega_n_I);
	if (dev_omega_n_Q)
		hipFree(dev_omega_n_Q);
	if (dev_phi_n)
		hipFree(dev_phi_n);
	if (dev_cos_value)
		hipFree(dev_cos_value);
	if (dev_sin_value)
		hipFree(dev_sin_value);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		return false;
	}
	return isSucceed;
}
